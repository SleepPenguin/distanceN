#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include "hip/hip_runtime_api.h"
#include "ticktock.h"

using Dvector = thrust::device_vector<double>;
using Hvector = thrust::host_vector<double>;

template <typename T>
std::ostream &operator<<(std::ostream &os, const thrust::host_vector<T> &h_vec)
{
    os << "[";
    for (size_t i = 0; i < h_vec.size(); ++i)
    {
        os << h_vec[i];
        if (i != h_vec.size() - 1)
        {
            os << ", ";
        }
    }
    os << "]";
    return os;
}

// cal core
struct distance
{
    int N{};
    double *pD_vec;

    distance(int n, Dvector &d_vec) : N(n), pD_vec(thrust::raw_pointer_cast(d_vec.data())) {}

    __device__ double operator()(const int &idx)
    {
        // idx: 1 -> N*N
        int i = idx / N;
        int j = idx % N;
        return ((pD_vec[i] - pD_vec[j]) * (pD_vec[i] - pD_vec[j]) +
                (pD_vec[i + N] - pD_vec[j + N]) * (pD_vec[i + N] - pD_vec[j + N]) +
                (pD_vec[i + 2 * N] - pD_vec[j + 2 * N]) * (pD_vec[i + 2 * N] - pD_vec[j + 2 * N]));
    }
};

int main()
{
    int N = 10000;

    // Generate 3N random numbers on device.
    thrust::default_random_engine rng(1332);
    thrust::uniform_real_distribution<double> distribute(-50.0, 50.0);
    Hvector h_vec(3 * N);
    thrust::generate(h_vec.begin(), h_vec.end(), [&]
                     { return distribute(rng); });

    // x0,x1,x2...,y0,y1...,z0,z1...,zN  3*N
    Dvector d_vec = h_vec;

    auto begin = thrust::counting_iterator<int>(0);
    auto end = thrust::counting_iterator<int>(N * N);
    Dvector d_dis2(N * N);

    TICK(distanceCore);

    thrust::transform(begin, end, d_dis2.begin(), distance(N, d_vec));

    checkCudaErrors(hipDeviceSynchronize()); // spend more time than core
    TOCK(distanceCore);

    // Hvector h_dis2 = d_dis2;

    // std::cout << h_vec << std::endl;
    // std::cout << h_dis2 << std::endl;
}